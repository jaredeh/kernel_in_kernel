#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define N 1000000

__global__ void vector_add_gpu(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

unsigned long long dtime_usec(unsigned long long start=0) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return ((tv.tv_sec * 1000000ULL) + tv.tv_usec) - start;
}

int main() {
    int *a = (int *) malloc(N * sizeof(int));
    int *b = (int *) malloc(N * sizeof(int));
    int *c = (int *) malloc(N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Warm-up kernel launch
    vector_add_gpu<<<1, 1>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Launch kernel
    vector_add_gpu<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Record stop event
    hipEventRecord(stop);

    // Synchronize to wait for the kernel to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Convert milliseconds to microseconds
    float microseconds = milliseconds * 1000;
    printf("GPU Duration: %f us\n", microseconds);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
